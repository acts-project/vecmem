/** VecMem project, part of the ACTS project (R&D line)
 *
 * (c) 2021 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Local include(s).
#include "vecmem/memory/cuda/direct_memory_manager.hpp"
#include "vecmem/utils/cuda_error_handling.hpp"

// CUDA include(s).
#include <hip/hip_runtime.h>

// System include(s).
#include <algorithm>
#include <cassert>
#include <stdexcept>

namespace vecmem { namespace cuda {

   direct_memory_manager::direct_memory_manager( vecmem::memory::memory_type type,
                                                 std::size_t sizeInBytes )
   : m_type( type ) {

      // Allocate the requested amount of memory.
      set_maximum_capacity( sizeInBytes, DEFAULT_DEVICE );
   }

   direct_memory_manager::~direct_memory_manager() {

      // Free all the (still available) allocated memory.
      // Ignore the errors from these calls. The destruction of this object
      // may happen after the CUDA runtime has already "shut down". Leading
      // to a (silent) failure from these calls.
      if( m_type == vecmem::memory::memory_type::HOST ) {
         for( device_memory& dev : m_memory ) {
            for( void* ptr : dev.m_ptrs ) {
               VECMEM_CUDA_ERROR_IGNORE( hipFree( ptr ) );
            }
         }
      } else {
         for( device_memory& dev : m_memory ) {
            for( void* ptr : dev.m_ptrs ) {
               VECMEM_CUDA_ERROR_IGNORE( hipHostFree( ptr ) );
            }
         }
      }
   }

   void direct_memory_manager::set_maximum_capacity( std::size_t sizeInBytes,
                                                     int device ) {

      // Get the object responsible for this device.
      device_memory& mem = get_device_memory( device );

      // Make sure that this is possible.
      hipDeviceProp_t prop;
      VECMEM_CUDA_ERROR_CHECK( hipGetDeviceProperties( &prop, device ) );
      if( prop.totalGlobalMem < sizeInBytes ) {
         throw std::bad_alloc();
      }
      return;
   }

   std::size_t direct_memory_manager::available_memory( int device ) const {

      // Get a valid device.
      get_device( device );

      // Get the information directly from CUDA.
      hipDeviceProp_t prop;
      VECMEM_CUDA_ERROR_CHECK( hipGetDeviceProperties( &prop, device ) );
      return prop.totalGlobalMem;
    }

   void* direct_memory_manager::allocate( std::size_t sizeInBytes,
                                          int device ) {

      // Get the object responsible for this device.
      device_memory& mem = get_device_memory( device );

      // Do the allocation.
      void* result = nullptr;
      VECMEM_CUDA_ERROR_CHECK( hipSetDevice( device ) );
      switch( m_type ) {
      case vecmem::memory::memory_type::DEVICE:
         VECMEM_CUDA_ERROR_CHECK( hipMalloc( &result, sizeInBytes ) );
         break;
      case vecmem::memory::memory_type::HOST:
         VECMEM_CUDA_ERROR_CHECK( hipHostMalloc( &result, sizeInBytes ) );
         break;
      case vecmem::memory::memory_type::MANAGED:
         VECMEM_CUDA_ERROR_CHECK( hipMallocManaged( &result, sizeInBytes ) );
         break;
      default:
         assert( false );
         break;
      }

      // Update the internal state of the memory manager.
      mem.m_ptrs.push_back( result );

      // Apparently everything is okay.
      return result;
   }

   void direct_memory_manager::deallocate( void* ptr ) {

      // Find which device this allocation was made on.
      auto itr = std::find_if( m_memory.begin(), m_memory.end(),
                               [ ptr ]( const device_memory& m ) {
                                  return ( std::find( m.m_ptrs.begin(),
                                                      m.m_ptrs.end(),
                                                      ptr ) != m.m_ptrs.end() );
                               } );
      if( itr == m_memory.end() ) {
         throw std::runtime_error( "Couldn't find allocation" );
      }

      // De-allocate the memory.
      if( m_type == vecmem::memory::memory_type::HOST ) {
         VECMEM_CUDA_ERROR_CHECK( hipHostFree( ptr ) );
      } else {
         VECMEM_CUDA_ERROR_CHECK( hipFree( ptr ) );
      }

      // Forget about this allocation.
      auto ptr_itr = std::find( itr->m_ptrs.begin(), itr->m_ptrs.end(), ptr );
      if( ptr_itr == itr->m_ptrs.end() ) {
         throw std::runtime_error( "Internal logic error detected" );
      }
      itr->m_ptrs.erase( ptr_itr );
      return;
   }

   void direct_memory_manager::reset( int device ) {

      // Get the object responsible for this device.
      device_memory& mem = get_device_memory( device );

      // Deallocate all memory associated with the device.
      if( m_type == vecmem::memory::memory_type::HOST ) {
         for( void* ptr : mem.m_ptrs ) {
            VECMEM_CUDA_ERROR_CHECK( hipHostFree( ptr ) );
         }
      } else {
         for( void* ptr : mem.m_ptrs ) {
            VECMEM_CUDA_ERROR_CHECK( hipFree( ptr ) );
         }
      }
      mem.m_ptrs.clear();
      return;
   }

   bool direct_memory_manager::is_host_accessible() const {

      return ( m_type != vecmem::memory::memory_type::DEVICE );
   }

   void direct_memory_manager::get_device( int& device ) {

      // If the user didn't ask for a specific device, use the one currently
      // used by CUDA.
      if( device == DEFAULT_DEVICE ) {
         VECMEM_CUDA_ERROR_CHECK( hipGetDevice( &device ) );
      }
      return;
   }

   direct_memory_manager::device_memory&
   direct_memory_manager::get_device_memory( int& device ) {

      // Get a valid device.
      get_device( device );

      // Make sure that the internal storage variable is large enough.
      if( static_cast< std::size_t >( device ) >= m_memory.size() ) {
         m_memory.resize( device + 1 );
      }

      // Return the requested object.
      return m_memory[ device ];
   }

} } // namespace vecmem::cuda
