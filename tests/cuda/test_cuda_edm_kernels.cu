#include "hip/hip_runtime.h"
/* VecMem project, part of the ACTS project (R&D line)
 *
 * (c) 2023 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Local include(s).
#include "../common/jagged_soa_container_helpers.hpp"
#include "../common/simple_soa_container_helpers.hpp"
#include "test_cuda_edm_kernels.hpp"

// Project include(s).
#include "../../cuda/src/utils/cuda_error_handling.hpp"

__global__ void cudaSimpleFillKernel(
    vecmem::testing::simple_soa_container::view view) {

    // Get the thread index.
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Run the modification using the helper function.
    vecmem::testing::simple_soa_container::device device{view};
    if (i < device.capacity()) {
        vecmem::testing::simple_soa_container::device::size_type ii =
            device.push_back_default();
        vecmem::testing::fill(i, device);
    }
}

void cudaSimpleFill(vecmem::testing::simple_soa_container::view view) {

    // Launch the kernel.
    const unsigned int blockSize = 256;
    const unsigned int gridSize = (view.capacity() + blockSize - 1) / blockSize;
    cudaSimpleFillKernel<<<gridSize, blockSize>>>(view);

    // Check whether it succeeded to run.
    VECMEM_CUDA_ERROR_CHECK(hipGetLastError());
    VECMEM_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

__global__ void cudaJaggedFillKernel(
    vecmem::testing::jagged_soa_container::view view) {

    // Get the thread index.
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Run the modification using the helper function.
    vecmem::testing::jagged_soa_container::device device{view};
    vecmem::testing::fill(i, device);
}

void cudaJaggedFill(vecmem::testing::jagged_soa_container::view view) {

    // Launch the kernel.
    const unsigned int blockSize = 256;
    const unsigned int gridSize = (view.capacity() + blockSize - 1) / blockSize;
    cudaJaggedFillKernel<<<gridSize, blockSize>>>(view);

    // Check whether it succeeded to run.
    VECMEM_CUDA_ERROR_CHECK(hipGetLastError());
    VECMEM_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

__global__ void cudaSimpleModifyKernel(
    vecmem::testing::simple_soa_container::view view) {

    // Get the thread index.
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Run the modification using the helper function.
    vecmem::testing::simple_soa_container::device device{view};
    vecmem::testing::modify(i, device);
}

void cudaSimpleModify(vecmem::testing::simple_soa_container::view view) {

    // Launch the kernel.
    const unsigned int blockSize = 256;
    const unsigned int gridSize = (view.capacity() + blockSize - 1) / blockSize;
    cudaSimpleModifyKernel<<<gridSize, blockSize>>>(view);

    // Check whether it succeeded to run.
    VECMEM_CUDA_ERROR_CHECK(hipGetLastError());
    VECMEM_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

__global__ void cudaJaggedModifyKernel(
    vecmem::testing::jagged_soa_container::view view) {

    // Get the thread index.
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Run the modification using the helper function.
    vecmem::testing::jagged_soa_container::device device{view};
    vecmem::testing::modify(i, device);
}

void cudaJaggedModify(vecmem::testing::jagged_soa_container::view view) {

    // Launch the kernel.
    const unsigned int blockSize = 256;
    const unsigned int gridSize = (view.capacity() + blockSize - 1) / blockSize;
    cudaJaggedModifyKernel<<<gridSize, blockSize>>>(view);

    // Check whether it succeeded to run.
    VECMEM_CUDA_ERROR_CHECK(hipGetLastError());
    VECMEM_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}
