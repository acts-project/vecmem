#include "hip/hip_runtime.h"
/** VecMem project, part of the ACTS project (R&D line)
 *
 * (c) 2021 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Local include(s).
#include "../../cuda/src/utils/cuda_error_handling.hpp"
#include "../../cuda/src/utils/cuda_wrappers.hpp"
#include "test_cuda_containers_kernels.cuh"
#include "vecmem/containers/const_device_array.hpp"
#include "vecmem/containers/const_device_vector.hpp"
#include "vecmem/containers/device_vector.hpp"
#include "vecmem/containers/jagged_device_vector.hpp"
#include "vecmem/containers/static_array.hpp"
#include "vecmem/memory/atomic.hpp"

/// Kernel performing a linear transformation using the vector helper types
__global__ void linearTransformKernel(
    vecmem::data::vector_view<const int> constants,
    vecmem::data::vector_view<const int> input,
    vecmem::data::vector_view<int> output) {

    // Find the current index.
    const std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= input.size()) {
        return;
    }

    // Create the helper containers.
    const vecmem::const_device_array<int, 2> constantarray1(constants);
    const vecmem::static_array<int, 2> constantarray2 = {constantarray1[0],
                                                         constantarray1[1]};
    const vecmem::const_device_vector<int> inputvec(input);
    vecmem::device_vector<int> outputvec(output);

    // Perform the linear transformation.
    outputvec.at(i) =
        inputvec.at(i) * constantarray1.at(0) + vecmem::get<1>(constantarray2);
    return;
}

void linearTransform(vecmem::data::vector_view<const int> constants,
                     vecmem::data::vector_view<const int> input,
                     vecmem::data::vector_view<int> output) {

    // Launch the kernel.
    linearTransformKernel<<<1, input.size()>>>(constants, input, output);
    // Check whether it succeeded to run.
    VECMEM_CUDA_ERROR_CHECK(hipGetLastError());
    VECMEM_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

void linearTransform(vecmem::data::vector_view<const int> constants,
                     vecmem::data::vector_view<const int> input,
                     vecmem::data::vector_view<int> output,
                     const vecmem::cuda::stream_wrapper& stream) {

    // Launch the kernel.
    linearTransformKernel<<<1, input.size(), 0,
                            vecmem::cuda::details::get_stream(stream)>>>(
        constants, input, output);
    // Check whether it succeeded to launch.
    VECMEM_CUDA_ERROR_CHECK(hipGetLastError());
}

/// Kernel performing some basic atomic operations.
__global__ void atomicTransformKernel(std::size_t iterations,
                                      vecmem::data::vector_view<int> data) {

    // Find the current global index.
    const std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= (data.size() * iterations)) {
        return;
    }

    // Get a pointer to the integer that this thread will work on.
    const std::size_t array_index = i % data.size();
    assert(array_index < data.size());
    int* ptr = data.ptr() + array_index;

    // Do some simple stuff with it.
    vecmem::atomic<int> a(ptr);
    a.fetch_add(4);
    a.fetch_sub(2);
    a.fetch_and(0xffffffff);
    a.fetch_or(0x00000000);
    return;
}

void atomicTransform(unsigned int iterations,
                     vecmem::data::vector_view<int> vec) {

    // Launch the kernel.
    atomicTransformKernel<<<iterations, vec.size()>>>(iterations, vec);
    // Check whether it succeeded to run.
    VECMEM_CUDA_ERROR_CHECK(hipGetLastError());
    VECMEM_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

/// Kernel filtering the input vector elements into the output vector
__global__ void filterTransformKernel(
    vecmem::data::vector_view<const int> input,
    vecmem::data::vector_view<int> output) {

    // Find the current global index.
    const std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= input.size()) {
        return;
    }

    // Set up the vector objects.
    const vecmem::const_device_vector<int> inputvec(input);
    vecmem::device_vector<int> outputvec(output);

    // Add this thread's element, if it passes the selection.
    const int element = inputvec.at(i);
    if (element > 10) {
        outputvec.push_back(element);
    }
    return;
}

void filterTransform(vecmem::data::vector_view<const int> input,
                     vecmem::data::vector_view<int> output) {

    // Launch the kernel.
    filterTransformKernel<<<1, input.size()>>>(input, output);
    // Check whether it succeeded to run.
    VECMEM_CUDA_ERROR_CHECK(hipGetLastError());
    VECMEM_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

/// Kernel filtering the input vector elements into the output vector
__global__ void filterTransformKernel(
    vecmem::data::jagged_vector_view<const int> input,
    vecmem::data::jagged_vector_view<int> output) {

    // Find the current indices.
    const std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= input.m_size) {
        return;
    }
    const std::size_t j = blockIdx.y * blockDim.y + threadIdx.y;
    if (j >= input.m_ptr[i].size()) {
        return;
    }

    // Set up the vector objects.
    const vecmem::jagged_device_vector<const int> inputvec(input);
    vecmem::jagged_device_vector<int> outputvec(output);

    // Keep just the odd elements.
    const int value = inputvec[i][j];
    if ((value % 2) != 0) {
        outputvec.at(i).push_back(value);
    }
    return;
}

void filterTransform(vecmem::data::jagged_vector_view<const int> input,
                     unsigned int max_vec_size,
                     vecmem::data::jagged_vector_view<int> output) {

    // Launch the kernel.
    dim3 dimensions(static_cast<unsigned int>(input.m_size), max_vec_size);
    filterTransformKernel<<<1, dimensions>>>(input, output);
    // Check whether it succeeded to run.
    VECMEM_CUDA_ERROR_CHECK(hipGetLastError());
    VECMEM_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

/// Kernel filling a jagged vector to its capacity
__global__ void fillTransformKernel(
    vecmem::data::jagged_vector_view<int> vec_data) {

    // Find the current index.
    const std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= vec_data.m_size) {
        return;
    }

    // Create a device vector on top of the view.
    vecmem::jagged_device_vector<int> vec(vec_data);

    // Fill the vectors to their capacity.
    while (vec[i].size() < vec[i].capacity()) {
        vec[i].push_back(1);
    }
}

void fillTransform(vecmem::data::jagged_vector_view<int> vec) {

    // Launch the kernel
    fillTransformKernel<<<static_cast<unsigned int>(vec.m_size), 1>>>(vec);

    // Check whether it succeeded to run.
    VECMEM_CUDA_ERROR_CHECK(hipGetLastError());
    VECMEM_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

__global__ void readArrayKernel(
    vecmem::static_array<vecmem::data::vector_view<int>, 3> arr_vec) {

    vecmem::device_vector<int> vec(arr_vec[0]);

    // It's OK
    printf("%d", vec.size());

    // this doesn't work
    printf("%d", vec[0]);
}

void readArray(
    vecmem::static_array<vecmem::data::vector_view<int>, 3> arr_vec) {

    readArrayKernel<<<1, 1>>>(arr_vec);

    // Check whether it succeeded to run.
    VECMEM_CUDA_ERROR_CHECK(hipGetLastError());
    VECMEM_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}